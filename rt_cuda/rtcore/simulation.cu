#include "hip/hip_runtime.h"
//#include <iostream>
#include "simulation.h"
#include "advance_beam.h"
#include "streamer.h"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

/*
 * A quick function to check if there was an error with the last Kernel Call.
 * Fill in the char * with a descriptive error message. This function terminates
 * the program.
 */
void check_call(const char *func){
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess){
        printf("%s: %s\n",func,hipGetErrorString(error));
        exit(1);
    }   
}

/*
 * This destructor copies all of the data back from the GPU to the CPU. 
 * Then all of the memory on the GPU is cleaned up.
 */
Simulation::~Simulation() {

    //Streamers_free(this->oStreamers);

    int nRay = this->nRay;
    int nRay3 = this->nRay*3;
    int rtmode = this->rtmode;
    int nIter = this->nIter;
    int nTracedPts = this->nTracedPts;

    // printf("~Simulation:: before hipMemcpyDeviceToHost\n");

    /* Copy arrays back from device to host memory */

    CUDA_CALL(hipMemcpy(this->Pos_ID_h, this->Pos_ID, sizeof(double)*nRay3,
			 DtoH));
    CUDA_CALL(hipMemcpy(this->Dir_ID_h, this->Dir_ID, sizeof(double)*nRay3,
			 DtoH));
    CUDA_CALL(hipMemcpy(this->DS_I_h, this->DS_I, sizeof(double)*nRay,
			 DtoH));
    CUDA_CALL(hipMemcpy(this->Flags_I_h, this->Flags_I,
			 sizeof(short)*nRay, DtoH));

    if(rtmode == 2)
	CUDA_CALL(hipMemcpy(this->Tbr_I_h, this->Tbr_I, sizeof(double)*nRay,
			     DtoH));
    if(rtmode == 3){
	CUDA_CALL(hipMemcpy(this->TbrIQUV_IP_h, this->TbrIQUV_IP, 
			     sizeof(double)*nRay*4, DtoH));
	CUDA_CALL(hipMemcpy(this->Bfield_ID_h, this->Bfield_ID,
			     sizeof(double)*nRay3, DtoH)); }
    if(rtmode == 2 || rtmode == 3)
	CUDA_CALL(hipMemcpy(this->OpDepth_I_h, this->OpDepth_I,
			     sizeof(double)*nRay, DtoH));

    CUDA_CALL(hipMemcpy(this->Rho_I_h, this->Rho_I, sizeof(double)*nRay,
			 DtoH));
    CUDA_CALL(hipMemcpy(this->GradRho_ID_h, this->GradRho_ID,
			 sizeof(double)*nRay3, DtoH));
    CUDA_CALL(hipMemcpy(this->PosPr_ID_h, this->PosPr_ID, sizeof(double)*nRay3,
			 DtoH));
    CUDA_CALL(hipMemcpy(this->DirPr_ID_h, this->DirPr_ID,
			 sizeof(double)*nRay3, DtoH));
    CUDA_CALL(hipMemcpy(this->DS_New_I_h, this->DS_New_I,
			 sizeof(double)*nRay, DtoH));
    CUDA_CALL(hipMemcpy(this->DistToCrSurf_I_h, this->DistToCrSurf_I,
			 sizeof(double)*nRay, DtoH));

    if(nTracedPts > 0) {
	/* Traced Rays */
	// printf("~Simulation:: before hipMemcpy(TracedPts_I_h)\n");
	CUDA_CALL(hipMemcpy(this->TracedPts_I_h, this->TracedPts_I,
			     sizeof(int)*nTracedPts, DtoH));
	CUDA_CALL(hipMemcpy(this->Trajectories_I_h, this->Trajectories_I,
			     sizeof(double)*nTracedPts*nIter*3, 
			     DtoH));
	CUDA_CALL(hipMemcpy(this->LastStep_I_h, this->LastStep_I,
			     sizeof(int)*nTracedPts, DtoH));
	CUDA_CALL(hipMemcpy(this->idxTracedPts_I_h, this->idxTracedPts_I,
			     sizeof(int)*nRay, DtoH));
	/* End Traced Rays */
    }


    /* Free device memory */

    CUDA_CALL(hipFree(this->prm));
    CUDA_CALL(hipFree(this->Pos_ID));
    CUDA_CALL(hipFree(this->Dir_ID));
    CUDA_CALL(hipFree(this->DS_I));
    CUDA_CALL(hipFree(this->Flags_I));

    if(rtmode == 2)
	CUDA_CALL(hipFree(this->Tbr_I));
    if(rtmode == 3) {
	CUDA_CALL(hipFree(this->TbrIQUV_IP));
	CUDA_CALL(hipFree(this->Bfield_ID)); }

    if(rtmode == 2 || rtmode == 3)
	CUDA_CALL(hipFree(this->OpDepth_I));

    CUDA_CALL(hipFree(this->Rho_I));
    CUDA_CALL(hipFree(this->GradRho_ID));
    CUDA_CALL(hipFree(this->PosPr_ID));
    CUDA_CALL(hipFree(this->DirPr_ID));
    CUDA_CALL(hipFree(this->DS_New_I));
    CUDA_CALL(hipFree(this->DistToCrSurf_I));

    if(nTracedPts > 0) {
	CUDA_CALL(hipFree(this->TracedPts_I));
	CUDA_CALL(hipFree(this->Trajectories_I));
	CUDA_CALL(hipFree(this->LastStep_I));
	CUDA_CALL(hipFree(this->idxTracedPts_I));
    }

    free(this->idxTracedPts_I_h); /* Free HOST memory from idxTracedPts_I_h */
  
}   /* Simulation::~Simulation()   */

/*
 * In the constructor, a copy of all the parameter fields is made on the device.
 * Therefore there are two copies of each variable stored. The normal fields
 * accessed by sim->field are the device copies. Fields accessed by
 * sim_h->field_h are the fields on the CPU
 */
Simulation::Simulation(
		       int nIter,
		       int nRay,
		       int nTracedPts,
		       struct param *prm,
		       double *Pos_ID, 
		       double *Dir_ID, 
		       double *DS_I, 
		       short *Flags_I,
		       double *Tbr_I, 
		       double *TbrIQUV_IP, 
		       double *OpDepth_I,
		       double *Rho_I,
		       double *GradRho_ID,
		       double *Bfield_ID,
		       double *PosPr_ID,
		       double *DirPr_ID,
		       double *DS_New_I,
		       double *DistToCrSurf_I,
		       int    *TracedPts_I,
		       double *Trajectories_I,
		       int    *LastStep_I,
		       int rtmode,
		       int scattering,
		       double rsph,
		       double *theta,
		       double *phi,
		       double *orientation,
		       double *density,
		       double *baseStrength,
		       double *stalkStrength,
		       double *scale) {


    //printf("Simulation: before calling Streamers::makeStreamer()\n");


    // this->oStreamers = Streamers_new();        

    // int i = 0;

    // while (!isnan(theta[i])) {
    //   Streamers_makeStreamer(this->oStreamers,
    // 			   1.0,
    // 			   theta[i],
    // 			   phi[i],
    // 			   orientation[i],
    // 			   density[i],
    // 			   baseStrength[i],
    // 			   stalkStrength[i],
    // 			   scale[i],
    // 			   scale[i],
    // 			   scale[i]);
    //   i++;
    // } 

    /*===================================================================
     *               Internal array nitializations
     *===================================================================*/
    int iRay, itrj;

    idxTracedPts_I = (int *) malloc(nRay*sizeof(int));

    for (int i=0; i<nRay; i++) idxTracedPts_I[i] = 0;

    for (int i=0; i<nTracedPts; i++) {
	iRay = TracedPts_I[i];
	idxTracedPts_I[iRay] = (nIter*i + 0)*3 + 0;  // nIter*iRay*3; 
    }

    this->nIter = nIter;
    this->nRay = nRay;
    this->nTracedPts=nTracedPts;
    this->prm_h = prm;
    this->Pos_ID_h = Pos_ID; 
    this->Dir_ID_h = Dir_ID; 
    this->DS_I_h = DS_I; 
    this->Flags_I_h = Flags_I;
    this->Tbr_I_h = Tbr_I; 
    this->TbrIQUV_IP_h = TbrIQUV_IP; 
    this->OpDepth_I_h = OpDepth_I;
    this->Rho_I_h = Rho_I;
    this->GradRho_ID_h = GradRho_ID;
    this->Bfield_ID_h = Bfield_ID;
    this->PosPr_ID_h = PosPr_ID;
    this->DirPr_ID_h = DirPr_ID;
    this->DS_New_I_h = DS_New_I;
    this->DistToCrSurf_I_h = DistToCrSurf_I;
    this->TracedPts_I_h = TracedPts_I;
    this->idxTracedPts_I_h = idxTracedPts_I;
    this->Trajectories_I_h = Trajectories_I; /* Fixed ERROR: Was X_I = X_I_h; */
    this->LastStep_I_h = LastStep_I;         /* Fixed ERROR: Was X_I = X_I_h; */
    this->rtmode = rtmode;
    this->scattering = scattering;
    this->rsph = rsph;

    int nRay3 = nRay*3;


    /* Allocate device memory for arrays */

    CUDA_CALL(hipMalloc((void **) &this->prm, sizeof(struct param)));
    CUDA_CALL(hipMalloc((void **) &this->Pos_ID, sizeof(double)*nRay3));
    CUDA_CALL(hipMalloc((void **) &this->Dir_ID, sizeof(double)*nRay3));
    CUDA_CALL(hipMalloc((void **) &this->DS_I, sizeof(double)*nRay));
    CUDA_CALL(hipMalloc((void **) &this->Flags_I, sizeof(short)*nRay));

    if(rtmode == 2)
	CUDA_CALL(hipMalloc((void **) &this->Tbr_I, sizeof(double)*nRay));
    if(rtmode == 3){
	CUDA_CALL(hipMalloc((void **) &this->TbrIQUV_IP, 
			     sizeof(double)*nRay*4));
	CUDA_CALL(hipMalloc((void **) &this->Bfield_ID, 
			     sizeof(double)*nRay3)); }
    if(rtmode == 2 || rtmode == 3)
	CUDA_CALL(hipMalloc((void **) &this->OpDepth_I, sizeof(double)*nRay));

    CUDA_CALL(hipMalloc((void **) &this->Rho_I,      sizeof(double)*nRay));
    CUDA_CALL(hipMalloc((void **) &this->GradRho_ID, sizeof(double)*nRay3));
    CUDA_CALL(hipMalloc((void **) &this->PosPr_ID,   sizeof(double)*nRay3));
    CUDA_CALL(hipMalloc((void **) &this->DirPr_ID,   sizeof(double)*nRay3));
    CUDA_CALL(hipMalloc((void **) &this->DS_New_I,   sizeof(double)*nRay));
    CUDA_CALL(hipMalloc((void **) &this->DistToCrSurf_I, sizeof(double)*nRay));

    if(nTracedPts > 0) {
	/* Traced Rays */
	CUDA_CALL(hipMalloc((void **) &this->TracedPts_I, 
			     sizeof(int)*nTracedPts));
	CUDA_CALL(hipMalloc((void **) &this->Trajectories_I,
			     sizeof(double)*nTracedPts*nIter*3));
	CUDA_CALL(hipMalloc((void **) &this->LastStep_I,
			     sizeof(int)*nTracedPts));
	CUDA_CALL(hipMalloc((void **) &this->idxTracedPts_I,
			     sizeof(int)*nRay));
	/* End Traced Rays */
    }


    /* Copy arrays to device memory */

    // printf("Simulation: before Copy arrays to device memory\n");

    CUDA_CALL(hipMemcpy(this->prm, this->prm_h, sizeof(struct param), HtoD));
    CUDA_CALL(hipMemcpy(this->Pos_ID, this->Pos_ID_h, 
			 sizeof(double)*nRay3, HtoD));
    CUDA_CALL(hipMemcpy(this->Dir_ID, this->Dir_ID_h,
			 sizeof(double)*nRay3, HtoD));
    CUDA_CALL(hipMemcpy(this->DS_I, this->DS_I_h, sizeof(double)*nRay, HtoD));
    CUDA_CALL(hipMemcpy(this->Flags_I, this->Flags_I_h,
			 sizeof(short)*nRay, HtoD));

    if(rtmode == 2)
	CUDA_CALL(hipMemcpy(this->Tbr_I, this->Tbr_I_h,
			     sizeof(double)*nRay, HtoD));
    if(rtmode == 3) {
	CUDA_CALL(hipMemcpy(this->TbrIQUV_IP, this->TbrIQUV_IP_h, 
			     sizeof(double*)*nRay*4, HtoD));
	CUDA_CALL(hipMemcpy(this->Bfield_ID, this->Bfield_ID_h,
			     sizeof(double)*nRay3, HtoD)); }
    if(rtmode == 2 || rtmode == 3)
	CUDA_CALL(hipMemcpy(this->OpDepth_I, this->OpDepth_I_h,
			     sizeof(double)*nRay, HtoD));

    CUDA_CALL(hipMemcpy(this->Rho_I, this->Rho_I_h,
			 sizeof(double)*nRay, HtoD));
    CUDA_CALL(hipMemcpy(this->GradRho_ID, this->GradRho_ID_h,
			 sizeof(double)*nRay3, HtoD));
    CUDA_CALL(hipMemcpy(this->PosPr_ID, this->PosPr_ID_h,
			 sizeof(double)*nRay3, HtoD));
    CUDA_CALL(hipMemcpy(this->DirPr_ID, this->DirPr_ID_h, 
			 sizeof(double)*nRay3, HtoD));
    CUDA_CALL(hipMemcpy(this->DS_New_I, this->DS_New_I_h,
			 sizeof(double)*nRay, HtoD));
    CUDA_CALL(hipMemcpy(this->DistToCrSurf_I, this->DistToCrSurf_I_h,
			 sizeof(double)*nRay, HtoD));

    if(nTracedPts > 0) {
	/* Traced Rays */
	CUDA_CALL(hipMemcpy(this->TracedPts_I, this->TracedPts_I_h,
			     sizeof(int)*nTracedPts, HtoD));
	CUDA_CALL(hipMemcpy(this->Trajectories_I, this->Trajectories_I_h,
			     sizeof(double)*nTracedPts*nIter*3, 
			     HtoD));
	CUDA_CALL(hipMemcpy(this->LastStep_I, this->LastStep_I_h,
			     sizeof(int)*this->nTracedPts, HtoD));
	CUDA_CALL(hipMemcpy(this->idxTracedPts_I, this->idxTracedPts_I_h,
			     sizeof(int)*nRay, HtoD));
	/* End Traced Rays */
    }

}

/*
 * This is the interface between C and C++ functions. The C function calls this
 * which then creates the necessary simulation object and calls the methods to
 * run the simulation.
 */
extern "C" void make_simulation(
				int nIter,
				int nRay,
				int nTracedPts,
				struct param *prm,
				double *Pos_ID, 
				double *Dir_ID, 
				double *DS_I, 
				short *Flags_I,
				double *Tbr_I, 
				double *TbrIQUV_IP, 
				double *OpDepth_I,
				double *Rho_I,
				double *GradRho_ID,
				double *Bfield_ID,
				double *PosPr_ID,
				double *DirPr_ID,
				double *DS_New_I,
				double *DistToCrSurf_I,
				int    *TracedPts_I,
				double *Trajectories_I,
				int    *LastStep_I,
				int rtmode,
				int scattering,
				double rsph,
				double *theta,
				double *phi,
				double *orientation,
				double *density,
				double *baseStrength,
				double *stalkStrength,
				double *scale){


    // printf("make_simulation: before creating Simulation sim()\n");


    Simulation sim (nIter,
                    nRay,
		    nTracedPts,
                    prm,
                    Pos_ID,
                    Dir_ID,
                    DS_I,
                    Flags_I,
                    Tbr_I,
                    TbrIQUV_IP,
                    OpDepth_I,
                    Rho_I,
                    GradRho_ID,
                    Bfield_ID,
                    PosPr_ID,
                    DirPr_ID,
                    DS_New_I,
                    DistToCrSurf_I,
		    TracedPts_I,
	            Trajectories_I,
	            LastStep_I,
                    rtmode,
                    scattering,
                    rsph,
                    theta,
                    phi,
                    orientation,
                    density,
                    baseStrength,
                    stalkStrength,
                    scale);





    sim.trace();

}
/*
 * This is the function that calls the advance_beam function. It makes a copy of
 * the simulation object onto the GPU and then calls advance_beam with a
 * reference to that GPU copy of simulation.
 */
void Simulation::trace() {
    Simulation *device_sim;

    /*
     * Constructor of the Simulation object allocates GPU memory for the 
     * arrays and copies their content to the GPU memory. The code below
     * allocates and copies to GPU the Simulation object itself, here 
     * device_sim.
     */
    CUDA_CALL(hipMalloc((void **) &device_sim, sizeof(Simulation)));
    CUDA_CALL(hipMemcpy(device_sim, this, sizeof(Simulation), HtoD));

    advance_beam(this, device_sim);

    CUDA_CALL(hipMemcpy(this, device_sim, sizeof(Simulation), DtoH));
    CUDA_CALL(hipFree(device_sim));

}


